#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

struct VertexInput {
  float3 position;
  float2 texCoord;
};

struct VertexOutput {
  float2 uv;
  float4 position;
};

struct FragmentInput {
  float2 uv;
};

struct FragmentOutput {
  float4 color;
};

__device__ VertexOutput main(VertexInput input) {
  VertexOutput output;
  return ['output'];
}

__device__ FragmentOutput main(FragmentInput input) {
  FragmentOutput output;
  float r;
  float g;
  float b;
  return ['output'];
}
