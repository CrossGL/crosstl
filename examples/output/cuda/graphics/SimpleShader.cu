#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

struct VertexInput {
  float3 position;
  float2 texCoord;
};

struct VertexOutput {
  float2 uv;
  float4 position;
};

struct FragmentInput {
  float2 uv;
};

struct FragmentOutput {
  float4 color;
};

__device__ VertexOutput main(VertexInput input) {
  VertexOutput output;
  output.uv = input.texCoord;
  output.position = vec4(input.position, 1.0);
  return output;
}

__device__ FragmentOutput main(FragmentInput input) {
  FragmentOutput output;
  float r;
  float g;
  float b;
  output.color = vec4(r, g, b, 1.0);
  return output;
}
